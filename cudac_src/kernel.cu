#include "hip/hip_runtime.h"
// source: https://github.com/sunnlo/BellmanFord/blob/master/cuda_bellman_ford.cu
#include <stdio.h>
#include "Coord2D.h"

__global__ void bellman_ford_kernel(int n, int* d_edges, int* d_distances, bool* next) {
    unsigned INF = 1000000;
    int thread = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // range check
    if(thread >= n) {return;}

    // every edge in d_edges is updated by a thread
    for(int i = 0; i < n; ++i) {
        for(int t = thread; t < n; t += stride) {
            int weight = d_edges[i*n+t];
            if(weight < INF) {
                int new_dist = d_distances[i] + weight;

                // update distance at t if new min found
                if(new_dist < d_distances[t]) {
                    d_distances[t] = new_dist;
                    *next = true; // return true through pointer param
                }
            }
        }
    }
}

/**
    @param blocksPerGrid
    @param theadsPerBlock
    @param n - number of vertices
    @param edges - (n*n)-sized array of edges; for a graph with n vertices, there are a maximum of n*(n-1) edges
    @param distances - array of distances from source vertex to other vertices in graph
**/
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int* edges, int* distances) {
    unsigned INF = 1000000;
    dim3 dim_grid(blocksPerGrid);
    dim3 dim_block(threadsPerBlock);

    int* d_edges;
    int* d_distances;

    // denote whether there should be another iteration of bellman-ford kernel
    bool* d_next; 
    bool* h_next; 

    // Allocate device variables
    hipMalloc(&d_edges, sizeof(int)*n*n);
    hipMalloc(&d_distances, sizeof(int)*n);
    hipMalloc(&d_next, sizeof(bool));

    // initialize distances to source to INF
    for(int i = 0; i < n; ++i) {
        distances[i] = INF;
    }
    distances[0] = 0; // dist from source to itself

    // Copy host variables to device
    hipMemcpy(d_edges, edges, sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(d_distances, distances, sizeof(int)*n, hipMemcpyHostToDevice);

    for(;;) {
        // initialize next iteration to false
        *h_next = false;
        hipMemcpy(d_next, h_next, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_kernel<<<dim_grid, dim_block>>> (n, d_edges, d_distances, d_next);
        hipDeviceSynchronize();

        // Copy result of next back to host
        hipMemcpy(h_next, d_next, sizeof(bool), hipMemcpyDeviceToHost);
        if(!(*h_next)) {
            break;
        }
    }
    // Copy distances array back to host
    hipMemcpy(distances, d_distances, sizeof(int)*n, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_edges);
    hipFree(d_distances);
    hipFree(d_next);
}


 __global__ void calcPaths(Coord2D* routes, int start, int end, int index, Coord2D* srcs, Coord2D* dests){

    int xStep = dests[index].first - srcs[index].first;
    int yStep = dests[index].second - srcs[index].second;

    if(xStep < 0) {xStep /= (-1)*xStep;}
    else if(xStep > 0) {xStep /= xStep;}
    if(yStep < 0) {yStep /= (-1)*yStep;}
    else if(yStep > 0) {yStep /= yStep;}
    routes[start].first = xStep;
    routes[start].second = yStep;
    printf("Routes at start: first=%d second=%d\n", routes[start].first, routes[start].second);
}

__global__ void getPaths(Coord2D* routes, int* sizes, Coord2D* srcs, Coord2D* dests){
    int tid = threadIdx.x + blockIdx.x*blockDim.x; //Num threads path_sz - 1
    printf("Inside kernel with thread: %d\n", tid);
    int start = sizes[tid];
    int end = sizes[tid+1];
    calcPaths<<<1, 1>>> (routes, start, end, tid, srcs, dests);
    hipDeviceSynchronize();
}
