#include "hip/hip_runtime.h"
// source: https://github.com/sunnlo/BellmanFord/blob/master/cuda_bellman_ford.cu
#include <stdio.h>
#include "Coord2D.h"

__global__ void bellman_ford_kernel(int n, int* d_edges, int* d_distances, bool* next) {
    unsigned INF = 1000000;
    int thread = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // range check
    if(thread >= n) {return;}

    // every edge in d_edges is updated by a thread
    for(int i = 0; i < n; ++i) {
        for(int t = thread; t < n; t += stride) {
            int weight = d_edges[i*n+t];
            if(weight < INF) {
                int new_dist = d_distances[i] + weight;

                // update distance at t if new min found
                if(new_dist < d_distances[t]) {
                    d_distances[t] = new_dist;
                    *next = true; // return true through pointer param
                }
            }
        }
    }
}

/**
    @param blocksPerGrid
    @param theadsPerBlock
    @param n - number of vertices
    @param edges - (n*n)-sized array of edges; for a graph with n vertices, there are a maximum of n*(n-1) edges
    @param distances - array of distances from source vertex to other vertices in graph
**/
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int* edges, int* distances) {
    unsigned INF = 1000000;
    dim3 dim_grid(blocksPerGrid);
    dim3 dim_block(threadsPerBlock);

    int* d_edges;
    int* d_distances;

    // denote whether there should be another iteration of bellman-ford kernel
    bool* d_next; 
    bool* h_next; 

    // Allocate device variables
    hipMalloc(&d_edges, sizeof(int)*n*n);
    hipMalloc(&d_distances, sizeof(int)*n);
    hipMalloc(&d_next, sizeof(bool));

    // initialize distances to source to INF
    for(int i = 0; i < n; ++i) {
        distances[i] = INF;
    }
    distances[0] = 0; // dist from source to itself

    // Copy host variables to device
    hipMemcpy(d_edges, edges, sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(d_distances, distances, sizeof(int)*n, hipMemcpyHostToDevice);

    for(;;) {
        // initialize next iteration to false
        *h_next = false;
        hipMemcpy(d_next, h_next, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_kernel<<<dim_grid, dim_block>>> (n, d_edges, d_distances, d_next);
        hipDeviceSynchronize();

        // Copy result of next back to host
        hipMemcpy(h_next, d_next, sizeof(bool), hipMemcpyDeviceToHost);
        if(!(*h_next)) {
            break;
        }
    }
    // Copy distances array back to host
    hipMemcpy(distances, d_distances, sizeof(int)*n, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_edges);
    hipFree(d_distances);
    hipFree(d_next);
}


__device__ int calcStep(int n1, int n2){
     int result = n2 - n1;
     if(result < 0){result = -1;}
     else if(result > 0){result = 1;}

    return result;
}

__global__ void getPaths(int totalSize, int* routesX, int* routesY, int* srcsX, int* srcsY, int* destsX, int* destsY, int* sizes){
    int tid = threadIdx.x + blockIdx.x*blockDim.x; //Num threads path_sz - 1
    printf("TID: %d --- srcsX: %d, srcsY: %d\n", tid, srcsX[tid], srcsY[tid]);
    printf("TID: %d --- destsX: %d, destsY: %d\n", tid, destsX[tid], destsY[tid]);
    printf("i: %d --- size %d\n", tid, sizes[tid]);
    int start;
    int end;
    int x = srcsX[tid];
    int y = srcsY[tid];
    int xBound = destsX[tid];
    int yBound = destsY[tid];
    int xStep = calcStep(x, xBound);
    int yStep = calcStep(y, yBound);
    int stepType = 0;
    if(tid == 0){
        start = 0;
        end = sizes[tid];
    }
    else{
        start = sizes[tid-1];
        end = sizes[tid];
    }
    
    for(int i = start; i < end; i++){
        stepType = i%2;
        routesX[i] = x;
	routesY[i] = y;
	if(stepType == 0){
	    if(x != xBound){
		x += xStep;
            }
            else if(y != yBound){
                y += yStep; 
            }
        }
        else{
            if(y != yBound){
                y += yStep;
            }
            else if (x != xBound){
                x += xStep;
            }
        }
           
    }
    __syncthreads();
}    
